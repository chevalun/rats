#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "time_test.h"

#define DIM 2048
#define TILE_DIM 16
#define BLOCK_ROWS 16

__global__ void transpose(int *new_m, int *old_m, int dim)
{
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

	int index_in  = xIndex + dim * yIndex;
	int index_out = yIndex + dim * xIndex;

	for(int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
		new_m[index_out + i] = old_m[index_in + i * dim];
}

int main()
{
	const int memSize = DIM * DIM * sizeof(int);
	int (*old_m)[DIM] = (int (*)[DIM]) malloc(memSize);
	int (*new_m)[DIM] = (int (*)[DIM]) malloc(memSize);
	
	for(int i = 0; i < DIM; ++i)
		for(int j = 0; j < DIM; ++j)
			old_m[i][j] = rand();
	
	START_TEST(10)
	
	dim3 grid(DIM/TILE_DIM, DIM/TILE_DIM), threads(TILE_DIM, BLOCK_ROWS);
	int *old_m_d, *new_m_d;
	hipMalloc((void **) &old_m_d, memSize);
	hipMalloc((void **) &new_m_d, memSize);
	hipMemcpy(old_m_d, &(old_m[0][0]), memSize, hipMemcpyHostToDevice);
	transpose<<<grid, threads>>>(new_m_d, old_m_d, DIM);
	hipMemcpy(new_m, new_m_d, memSize, hipMemcpyDeviceToHost);

	END_TEST
	
	return 0;
}
